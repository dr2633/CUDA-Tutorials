// Thread Hierarchy Example: Matrix Addition with Synchronization and Benchmarking
// Created by Derek Rosenzweig on 12/26/24

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

const int N = 1024;  // Matrix size (NxN)

// Kernel definition for matrix addition with synchronization
__global__ void MatAdd(float* A, float* B, float* C, int width) {
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Global thread ID in x-dimension
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Global thread ID in y-dimension

    // Allocate shared memory for cooperative data sharing within the block
    __shared__ float sharedA[16][16];
    __shared__ float sharedB[16][16];

    // Perform the addition if indices are within matrix bounds
    if (i < width && j < width) {
        int idx = j * width + i;  // Flatten 2D index

        // Load data into shared memory
        sharedA[threadIdx.y][threadIdx.x] = A[idx];
        sharedB[threadIdx.y][threadIdx.x] = B[idx];
        __syncthreads();  // Synchronize all threads in the block

        // Perform addition using shared memory
        C[idx] = sharedA[threadIdx.y][threadIdx.x] + sharedB[threadIdx.y][threadIdx.x];
    }
}

int main() {
    const int matrixSize = N * N;
    const int bytes = matrixSize * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);

    // Initialize host matrices
    for (int i = 0; i < matrixSize; i++) {
        h_A[i] = static_cast<float>(rand() % 100);
        h_B[i] = static_cast<float>(rand() % 100);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Define thread hierarchy
    dim3 threadsPerBlock(16, 16);  // 16x16 threads per block
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Benchmark kernel execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel execution time: %f ms\n", milliseconds);

    // Copy result back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < matrixSize; i++) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            printf("Error at index %d: %f\n", i, h_C[i]);
            return -1;
        }
    }
    printf("Matrix addition completed successfully.\n");

    // Clean up benchmark resources
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
